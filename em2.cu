
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <stdio.h>
#include <math.h>
#include <cstring>

using namespace std;


__global__ void compute_z(int *NOC_device,int *NOS_device,int *SC_device,float *a_device,float *b_device,float *Z_device,float *d_device){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < NOC_device[0]){
        float A = 0;
	float B = 0;
	for (int i = 0; i < NOS_device[0]; i++) {
  	    A = A + SC_device[i*NOC_device[0]+id] * log(a_device[i]) + (1-SC_device[i*NOC_device[0]+id]) * log(1 - a_device[i]);
	    B = B + SC_device[i*NOC_device[0]+id] * log(b_device[i]) + (1-SC_device[i*NOC_device[0]+id]) * log(1 - b_device[i]);
	    //printf("A[%d] is: %.10f  a[i]=%f B[%d] is: %.10f  b[i]=%f \n",i,log(a_device[i]),a_device[i],i,log(b_device[i]),b_device[i]);
        }
	A = exp(A);
	B = exp(B);
	Z_device[id] = (A * d_device[0]) / ((A * d_device[0]) + (B * (1 - d_device[0])));
	//printf("Z[%d] is: %.10f \n",id,Z_device[id]);
	//printf("A[%d] is: %.10f \n",id,log(a_device[i]);
	//printf("B[%d] is: %.10f \n",id,log(b_device[i]);	
	//printf("d[%d] is: %.10f \n",id,d_device[0]);
    } 
}


__global__ void compute_theta(int *NOC_device,int *NOS_device,int *SC_device,float *a_device,float *b_device,float *Z_device,float *d_device){
        int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id < NOS_device[0]){
	    float tempz = 0;
	    float totalz = 0;
	    int cnt = 0;
	    for (int j = 0; j < NOC_device[0]; j++) {
	        if (SC_device[id * NOC_device[0] +j] == 1) {
		    tempz = tempz + Z_device[j];
		    cnt = cnt + 1;
		}
  		totalz = totalz + Z_device[j];
	    }
	    a_device[id] = tempz / totalz;
	    b_device[id] = (cnt - tempz) / (NOC_device[0] - totalz);
	    d_device[0] = totalz / NOC_device[0];
	}    
}


int main() {


    FILE *input = fopen("TestSensingMatrix.txt", "r");

    const int NOS = 30;
    const int NOC = 2000;
    const int MAX_IT = 10;

    int *NOS_device;
    int *NOC_device;
    int *MAX_IT_device;
    hipMalloc(&NOS_device, sizeof(int));
    hipMalloc(&NOC_device, sizeof(int));
    hipMalloc(&MAX_IT_device, sizeof(int));
    hipMemcpy( NOS_device, &NOS, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( NOC_device, &NOC, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( MAX_IT_device, &MAX_IT, sizeof(int), hipMemcpyHostToDevice);

    hipError_t malloc_error_check = hipGetLastError();
    if(malloc_error_check != hipSuccess){
    
        printf("malloc_error_check: CUDA error: %s\n", hipGetErrorString(malloc_error_check));
        exit(-1);
    }

    int SC[NOS*NOC];
    std::memset(SC, 0, sizeof(SC));

    int *SC_device;
    hipMalloc(&SC_device, NOS*NOC*sizeof(int));
    hipMemset(SC_device, 0, sizeof(int)*NOS*NOC);

    // Generate the SC matrix
    int row[2];
    while (fscanf(input, "%d,%d", &row[0], &row[1]) == 2) {
        SC[(row[0] - 1)*NOC + (row[1] - 1)] = 1;
    }
    
    hipMemcpy(SC_device, SC, sizeof(int)*NOS*NOC, hipMemcpyHostToDevice); 
    
    hipError_t malloc_error_check2 = hipGetLastError();
    if(malloc_error_check2 != hipSuccess){
    
        printf("malloc_error_check2: CUDA error: %s\n", hipGetErrorString(malloc_error_check2));
        exit(-1);
    }

    // s
    float s[NOS];

    float *s_device;
    hipMalloc(&s_device, NOS*sizeof(float));

    std::memset(s, 0, sizeof(s));
    for (int x = 0; x < NOS; x++) {
        int cnt = 0;
        for (int y = 0; y < NOC; y++) {
            if (SC[x*NOC+y] == 1) {
                cnt = cnt + 1;
            }
        }
        s[x] = cnt * 1.0 / NOC;
    }

    hipMemcpy(s_device, s, sizeof(float)*NOS, hipMemcpyHostToDevice);

    // theta[ai]
    float a[NOS];
    float b[NOS];
    float d = 0.5;
    float *d_device;
    hipMalloc(&d_device, sizeof(float));
    hipMemcpy(d_device, &d, sizeof(float), hipMemcpyHostToDevice);

    float Z[NOC];

    std::memset(a, 0, sizeof(a));
    std::memset(b, 0, sizeof(b));
    std::memset(Z, 0, sizeof(Z));


    float *a_device;
    float *b_device;
    float *Z_device;
    hipMalloc(&a_device, NOS*sizeof(float));
    hipMalloc(&b_device, NOS*sizeof(float));
    hipMalloc(&Z_device, NOC*sizeof(float));
    hipMemset(Z_device, 0, sizeof(float)*NOC);
    hipMemset(a_device, 0, sizeof(float)*NOS);
    hipMemset(b_device, 0, sizeof(float)*NOS);

    hipError_t malloc_error_check3 = hipGetLastError();
    if(malloc_error_check3 != hipSuccess){
    
        printf("malloc_error_check3: CUDA error: %s\n", hipGetErrorString(malloc_error_check3));
        exit(-1);
    }

    for (int x = 0; x < NOS; x++) {
        a[x] = s[x];
        b[x] = 0.5 * s[x];
    }

    hipMemcpy(a_device, a, sizeof(float)*NOS, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b, sizeof(float)*NOS, hipMemcpyHostToDevice);
 

    hipError_t malloc_error_check4 = hipGetLastError();
    if(malloc_error_check4 != hipSuccess){

        printf("malloc_error_check4: CUDA error: %s\n", hipGetErrorString(malloc_error_check4));
        exit(-1);
    }
 
    dim3 grid_vertex((int)ceil((float)NOC/(float)1024),1), block_vertex(1024,1);
    dim3 grid_vertex2((int)ceil((float)NOS/(float)1024),1), block_vertex2(1024,1); 


    for (int itn = 0; itn < MAX_IT; itn++) {
        // Compute Z(t, j)
        compute_z<<<grid_vertex,block_vertex>>>(NOC_device,NOS_device,SC_device,a_device,b_device,Z_device,d_device);
        //for(int foo1=0;foo1<NOC;foo1++){
          //  printf("z[%d] = %f \n",foo1,z2[foo1]);
        //}
        hipError_t iter_error = hipGetLastError();
        if(iter_error != hipSuccess)
        {
            printf("iter_error: CUDA error: %s\n", hipGetErrorString(iter_error));
            exit(-1);
        }
	hipDeviceSynchronize();
        compute_theta<<<grid_vertex2,block_vertex2>>>(NOC_device,NOS_device,SC_device,a_device,b_device,Z_device,d_device);
	hipDeviceSynchronize();
    }

    hipError_t vertex_filter_errorri = hipGetLastError();
    if(vertex_filter_errorri != hipSuccess)
    {
        printf("FilterFrontierrri: CUDA error: %s\n", hipGetErrorString(vertex_filter_errorri));
        exit(-1);
    }

    // end of while
    compute_z<<<grid_vertex,block_vertex>>>(NOC_device,NOS_device,SC_device,a_device,b_device,Z_device,d_device);

    hipError_t vertex_filter_errorr2 = hipGetLastError();
    if(vertex_filter_errorr2 != hipSuccess)
    {
        printf("FilterFrontierrr2: CUDA error: %s\n", hipGetErrorString(vertex_filter_errorr2));
        exit(-1);
    }


    hipMemcpy(Z, Z_device, sizeof(float)*NOC, hipMemcpyDeviceToHost);
    hipMemcpy(a, a_device, sizeof(float)*NOS, hipMemcpyDeviceToHost);
    hipMemcpy(b, b_device, sizeof(float)*NOS, hipMemcpyDeviceToHost);
    hipMemcpy(&d, d_device, sizeof(float), hipMemcpyDeviceToHost);

    FILE *groundtruth = fopen("TestGroundTruth.txt", "r");
    int gt[NOC];
    std::memset(gt, 0, sizeof(gt));

    while (fscanf(input, "%d,%d", &row[0], &row[1]) == 2) {
        gt[row[0] - 1] = row[1];
    }

    int out[NOC];
    std::memset(out, 0, sizeof(out));
   
    FILE *output = fopen("outtie.txt", "w");
    for (int j = 0; j < NOC; j++) {
    	if (Z[j] >= 0.5) {
	    	out[j] = 1;
        }
    }
    float t[NOS];
    std::memset(t, 0, sizeof(t));

    for (int i = 0; i < NOS; i++) {
      t[i] = (a[i]*d) / ((a[i]*d) + (b[i]*(1-d)));
        cout << t[i] << endl;
    }

    
    for (int j = 0; j < NOC; j++) {
	    fprintf(output, "%d, %d\n", j+1, out[j]);
    }

    fclose(input); 
 
    //Free GPU Memory
    hipFree(NOS_device);
    hipFree(NOC_device);
    hipFree(MAX_IT_device);
    hipFree(SC_device);
    hipFree(s_device);
    hipFree(d_device);
    hipFree(a_device); 
    hipFree(b_device);
    hipFree(Z_device);  
}

